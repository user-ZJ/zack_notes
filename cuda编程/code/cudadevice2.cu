udaSetDevice(0); // Set device 0 as current

float *p0;
size t size = 1024 * sizeof(float) ;
hipMalloc(&p0, size); // Allocate memory on device 0
MyKernel<<<1000, 128>>>(p0); // Launch kernel on device 0
hipSetDevice(1); // Set device 1 as current
hipDeviceEnablePeerAccess(0, 0); // Enable peer-to-peer access with device 0
// Launch kernel on device 1
// This kernel launch can access memory on device 0 at address p0
MyKernel<<<1000, 128>>>(p0);