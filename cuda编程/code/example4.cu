#include "hip/hip_runtime.h"
// Host code
int width = 64, height = 64;

float devPtr;
size_t pitch;
hipMallocPitch(&devPtr, &pitch,width*sizeof(float) , height);
MyKernel<<<100, 512>>>(devPtr, pitch, width, height);
// Device code
__global__ void MyKernel(float *devPtr,size_t pitch, int width, int height){
    for ( int r = 0; r < height; ++r){
        float *row = (float *)((char *)devPtr + r*pitch);
        for ( int c = 0; c <width; ++c) {
            float element = row[c];
        }
    }
}