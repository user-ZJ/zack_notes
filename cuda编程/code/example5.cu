#include "hip/hip_runtime.h"
// Host code
int width = 64, height = 64, depth = 64;
hipExtent extent = make hipExtent(width*sizeof(float),height, depth);
hipPitchedPtr devPitchedPtr;
hipMalloc3D(&devPitchedPtr, extent);
MyKernel<<<100, 512>>>(devPitchedPtr, width, height, depth);

// Device code
__global__ void MyKernel(hipPitchedPtr devPitchedPtr,int width, int height, int depth){
    char *devPtr = devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch*height;
    for ( int z = 0; z < depth; ++z) {
        char *slice = devPtr + z*slicePitch;
        for ( int y = 0; y < height; ++y) {
            float *row = (float *)(slice + y*pitch);
            for ( int x = 0; x < width; ++x){
                float element = row[x];
            }
        }
    }
}
