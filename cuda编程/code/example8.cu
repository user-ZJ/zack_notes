cudaEvent t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start, 0);
for ( int i = 0; i < 2; ++i) {
    hipMemcpyAsync(inputDev + i*size, inputHost + i*size,
                        size , hipMemcpyHostToDevice, stream[i]);
    MyKernel<<<100, 512, 0, stream[i]>>>(outputDev + i*size, inputDev + i*size, size ) ;
    hipMemcpyAsync(outputHost + i*size, outputDev + i*size,
        size , hipMemcpyDeviceToHost, stream[i]);
}
hipEventRecord(stop, 0);
hipEventSynchronize(stop);

float elapsedTime;
hipEventElapsedTime(&elapsedTime, start, stop);

hipEventDestroy(start);
hipEventDestroy(stop);