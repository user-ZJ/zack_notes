void CUDART_CB MyCallback(void *data){
    printf ("Inside callback %d\n", (int)data);
}

for( int i = 0; i < 2; ++i) {
    hipMemcpyAsync(devPtrIn[i], hostPtr[i], size,hipMemcpyHostToDevice, stream[i]);
    MyKernel<<<100, 512, 0, stream[i]>>>(devPtrOut[i], devPtrIn[i],size);
    hipMemcpyAsync(hostPtr[i], devPtrOut[i], size,hipMemcpyDeviceToHost, stream[i]);
    hipStreamAddCallback(stream[i], MyCallback, (void*)i, 0);
}