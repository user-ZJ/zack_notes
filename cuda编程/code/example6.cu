cudaStream t stream[2];
for ( int i = 0; i < 2; ++i)
hipStreamCreate(&stream[i]);

float *hostPtr;
hipHostMalloc(&hostPtr, 2*size);
for ( int i = 0; i < 2; ++i) {
    hipMemcpyAsync(inputDevPtr + i*size, hostPtr + i*size,size , 
                                hipMemcpyHostToDevice, stream[i]);
    MyKernel <<<100, 512, 0, stream[i]>>>(outputDevPtr + i*size, inputDevPtr + i*size, size);
    hipMemcpyAsync(hostPtr + i*size, outputDevPtr + i*size,size ,
                                hipMemcpyDeviceToHost, stream[i]);
}
for ( int i = 0; i < 2; ++i)
    hipStreamDestroy(stream[i]);